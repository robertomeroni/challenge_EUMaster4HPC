#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <time.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "hipblas.h"
#include "kernels.cu"
#include "cuda_error_check.h"
#include "solution_check.h"

clock_t start, end;
double cpu_time_used;

bool read_matrix_from_file(const char * filename, double ** matrix_out, size_t * num_rows_out, size_t * num_cols_out)
{
    double * matrix;
    size_t num_rows;
    size_t num_cols;

    FILE * file = fopen(filename, "rb");
    if(file == nullptr)
    {
        fprintf(stderr, "Cannot open output file\n");
        return false;
    }

    fread(&num_rows, sizeof(size_t), 1, file);
    fread(&num_cols, sizeof(size_t), 1, file);
    matrix = new double[num_rows * num_cols];
    fread(matrix, sizeof(double), num_rows * num_cols, file);

    *matrix_out = matrix;
    *num_rows_out = num_rows;
    *num_cols_out = num_cols;

    fclose(file);

    return true;
}

bool write_matrix_to_file(const char * filename, const double * matrix, size_t num_rows, size_t num_cols)
{
    FILE * file = fopen(filename, "wb");
    if(file == nullptr)
    {
        fprintf(stderr, "Cannot open output file\n");
        return false;
    }

    fwrite(&num_rows, sizeof(size_t), 1, file);
    fwrite(&num_cols, sizeof(size_t), 1, file);
    fwrite(matrix, sizeof(double), num_rows * num_cols, file);

    fclose(file);

    return true;
}

void print_matrix(const double * matrix, size_t num_rows, size_t num_cols, FILE * file = stdout)
{
    fprintf(file, "%zu %zu\n", num_rows, num_cols);
    for(size_t r = 0; r < num_rows; r++)
    {
        for(size_t c = 0; c < num_cols; c++)
        {
            double val = matrix[r * num_cols + c];
            printf("%+6.3f ", val);
        }
        printf("\n");
    }
}


void conjugate_gradients(const double * A, const double * b, double * x, size_t size, int max_iters, double rel_error)
{   
    double alpha, beta, bb, rr, rr_new;
    double pAp;
    const double gemv_alpha = 1.0;
    const double gemv_beta = 0.0;
    double * d_A = new double[size * size];
    double * d_Ap = new double[size];
    double * d_x = new double[size];
    double * d_b = new double[size];
    double * d_r = new double[size];
    double * d_p = new double[size];
    double * d_bb = new double;
    double * d_rr_new = new double;
    double * d_pAp = new double;
    int num_iters;

    int numBlocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int numThreads = BLOCK_SIZE;
    
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // memory allocation on the GPU.
    cudaErrorCheck(hipMalloc((void**)&d_x, size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_b, size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_r, size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_p, size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_bb, sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_rr_new, sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_A, size * size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_Ap, size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_pAp, sizeof(double)));

    // memory copy from the CPU to the GPU, copy of matrix A is done in parallel with initialization and dot product.
    cudaErrorCheck(hipMemcpyAsync(d_A, A, size * size * sizeof(double), hipMemcpyHostToDevice, stream1));
    cudaErrorCheck(hipMemcpy(d_b, b, size * sizeof(double), hipMemcpyHostToDevice));
    
    dot <<<numBlocks, numThreads, 0>>> (d_b, d_b, d_bb, size);
    initialization <<<numBlocks, numThreads, 0, stream2>>> (d_x, d_b, d_r, d_p, size);
    cudaErrorCheck(hipMemcpy(&bb, d_bb, sizeof(double), hipMemcpyDeviceToHost));
    // wait for matrix A to be copied to the GPU
    cudaErrorCheck(hipDeviceSynchronize());
    rr = bb;

    for(num_iters = 1; num_iters <= max_iters; num_iters++)
    {
        hipblasDgemv(handle, HIPBLAS_OP_N, size, size, &gemv_alpha, d_A, size, d_p, 1, &gemv_beta, d_Ap, 1);
        dot <<<numBlocks, numThreads>>> (d_p, d_Ap, d_pAp, size);
        cudaErrorCheck(hipMemcpy(&pAp, d_pAp, sizeof(double), hipMemcpyDeviceToHost));
        alpha = rr / pAp;
        axpby <<<numBlocks, numThreads>>> (-alpha, d_Ap, 1.0, d_r, size); 
        hipMemsetAsync(d_pAp, 0, sizeof(double)); // reset dot product to zero, done in parallel with stream1
        hipMemsetAsync(d_rr_new, 0, sizeof(double));
        hipStreamSynchronize(stream1); // ensure that axbpy on x from the previous iteration has terminated
        axpby <<<numBlocks, numThreads, 0, stream1>>> (alpha, d_p, 1.0, d_x, size); // x is not needed until the next iteration and is only get called by this kernel
        dot <<<numBlocks, numThreads>>> (d_r, d_r, d_rr_new, size);
        cudaErrorCheck(hipMemcpy(&rr_new, d_rr_new, sizeof(double), hipMemcpyDeviceToHost));
        beta = rr_new / rr;
        axpby <<<numBlocks, numThreads>>> (1.0, d_r, beta, d_p, size); // this can be done after beta is calculated
        rr = rr_new;
        if(std::sqrt(rr / bb) < rel_error) { break; }
    }

    cudaErrorCheck(hipMemcpyAsync(x, d_x, size * sizeof(double), hipMemcpyDeviceToHost,stream1));

    // cleaning up
    cudaErrorCheck(hipFree(d_x));
    cudaErrorCheck(hipFree(d_b));
    cudaErrorCheck(hipFree(d_r));
    cudaErrorCheck(hipFree(d_p));
    cudaErrorCheck(hipFree(d_A));
    cudaErrorCheck(hipFree(d_Ap));
    cudaErrorCheck(hipFree(d_bb));
    cudaErrorCheck(hipFree(d_pAp));
    cudaErrorCheck(hipFree(d_rr_new));
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipblasDestroy(handle);

    if(num_iters <= max_iters)
    {
        printf("Converged in %d iterations, relative error is %e\n", num_iters, std::sqrt(rr / bb));
    }
    else
    {
        printf("Did not converge in %d iterations, relative error is %e\n", max_iters, std::sqrt(rr / bb));
    }
}





int main(int argc, char ** argv)
{
    printf("Usage: ./random_matrix input_file_matrix.bin input_file_rhs.bin output_file_sol.bin max_iters rel_error\n");
    printf("All parameters are optional and have default values\n");
    printf("\n");

    const char * input_file_matrix = "io/matrix.bin";
    const char * input_file_rhs = "io/rhs.bin";
    const char * output_file_sol = "io/sol.bin";
    int max_iters = 1000;
    double rel_error = 1e-9;

    if(argc > 1) input_file_matrix = argv[1];
    if(argc > 2) input_file_rhs = argv[2];
    if(argc > 3) output_file_sol = argv[3];
    if(argc > 4) max_iters = atoi(argv[4]);
    if(argc > 5) rel_error = atof(argv[5]);

    printf("Command line arguments:\n");
    printf("  input_file_matrix: %s\n", input_file_matrix);
    printf("  input_file_rhs:    %s\n", input_file_rhs);
    printf("  output_file_sol:   %s\n", output_file_sol);
    printf("  max_iters:         %d\n", max_iters);
    printf("  rel_error:         %e\n", rel_error);
    printf("\n");



    double * matrix;
    double * rhs;
    size_t size;

    {
        printf("Reading matrix from file ...\n");
        size_t matrix_rows;
        size_t matrix_cols;
        bool success_read_matrix = read_matrix_from_file(input_file_matrix, &matrix, &matrix_rows, &matrix_cols);
        if(!success_read_matrix)
        {
            fprintf(stderr, "Failed to read matrix\n");
            return 1;
        }
        printf("Done\n");
        printf("\n");

        printf("Reading right hand side from file ...\n");
        size_t rhs_rows;
        size_t rhs_cols;
        bool success_read_rhs = read_matrix_from_file(input_file_rhs, &rhs, &rhs_rows, &rhs_cols);
        if(!success_read_rhs)
        {
            fprintf(stderr, "Failed to read right hand side\n");
            return 2;
        }
        printf("Done\n");
        printf("\n");

        if(matrix_rows != matrix_cols)
        {
            fprintf(stderr, "Matrix has to be square\n");
            return 3;
        }
        if(rhs_rows != matrix_rows)
        {
            fprintf(stderr, "Size of right hand side does not match the matrix\n");
            return 4;
        }
        if(rhs_cols != 1)
        {
            fprintf(stderr, "Right hand side has to have just a single column\n");
            return 5;
        }

        size = matrix_rows;
    }

    printf("Solving the system ...\n");
    double * sol = new double[size];

    start = clock();
    conjugate_gradients(matrix, rhs, sol, size, max_iters, rel_error);
    end = clock();

    printf("Done\n");
    printf("\n");

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Execution time: %f seconds\n", cpu_time_used);
    printf("\n");

    printf("Writing solution to file ...\n");
    bool success_write_sol = write_matrix_to_file(output_file_sol, sol, size, 1);
    if(!success_write_sol)
    {
        fprintf(stderr, "Failed to save solution\n");
        return 6;
    }
    printf("Done\n");
    printf("\n");

    #ifdef APPROX
    print_approx_solution(sol, size, 1);
    #endif

    delete[] matrix;
    delete[] rhs;
    delete[] sol;

    printf("Finished successfully\n");

    return 0;
}
