#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <time.h>
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <iostream>
#include "hipblas.h"
#include "kernels.cu"
#include "cuda_error_check.h"
#include "solution_check.h"

clock_t start, end;
double cpu_time_used;

bool read_matrix_from_file(const char * filename, double ** matrix_out, size_t * num_rows_out, size_t * num_cols_out)
{
    double * matrix;
    size_t num_rows;
    size_t num_cols;

    FILE * file = fopen(filename, "rb");
    if(file == nullptr)
    {
        fprintf(stderr, "Cannot open output file\n");
        return false;
    }

    fread(&num_rows, sizeof(size_t), 1, file);
    fread(&num_cols, sizeof(size_t), 1, file);
    matrix = new double[num_rows * num_cols];
    fread(matrix, sizeof(double), num_rows * num_cols, file);

    *matrix_out = matrix;
    *num_rows_out = num_rows;
    *num_cols_out = num_cols;

    fclose(file);

    return true;
}

bool write_matrix_to_file(const char * filename, const double * matrix, size_t num_rows, size_t num_cols)
{
    FILE * file = fopen(filename, "wb");
    if(file == nullptr)
    {
        fprintf(stderr, "Cannot open output file\n");
        return false;
    }

    fwrite(&num_rows, sizeof(size_t), 1, file);
    fwrite(&num_cols, sizeof(size_t), 1, file);
    fwrite(matrix, sizeof(double), num_rows * num_cols, file);

    fclose(file);

    return true;
}

void print_matrix(const double * matrix, size_t num_rows, size_t num_cols, FILE * file = stdout)
{
    fprintf(file, "%zu %zu\n", num_rows, num_cols);
    for(size_t r = 0; r < num_rows; r++)
    {
        for(size_t c = 0; c < num_cols; c++)
        {
            double val = matrix[r * num_cols + c];
            printf("%+6.3f ", val);
        }
        printf("\n");
    }
}



void nccl_conjugate_gradients(const double * A, const double * b, double * x, size_t size, int max_iters, double rel_error)
{   
    // NCCL initialization
    const int numGPUs = 4;
    int gpus[numGPUs] = {0, 1, 2, 3}; 
    ncclComm_t comms[numGPUs];
    hipStream_t stream1[numGPUs];
    hipStream_t stream2[numGPUs];

    bool not_divisible = size % numGPUs != 0;
    size_t num_rows = size / numGPUs + (not_divisible);
    int unused_rows = num_rows * numGPUs - size;

    double alpha, beta, bb, rr, rr_new;
    double pAp;
    const double gemv_alpha = 1.0;
    const double gemv_beta = 0.0;
    double **d_p = (double **) malloc(numGPUs * sizeof(double *));
    double **d_A = (double **) malloc(numGPUs * sizeof(double *));
    double **d_Ap = (double **) malloc(numGPUs * sizeof(double *));
    double * d_diagA[numGPUs];
    double * d_x;
    double * d_b;
    double * d_r;
    double * d_z;
    double * d_bb;
    double * d_rr_new;
    double * d_pAp;
    int num_iters;

    int numBlocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int numThreads = BLOCK_SIZE;

    ncclErrorCheck(ncclCommInitAll(comms, numGPUs, gpus));

     // memory allocation on the GPU.
    cudaErrorCheck(hipMalloc((void**)&d_x, size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_b, size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_r, size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_z, size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_bb, sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_rr_new, sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_pAp, sizeof(double)));

    for(int i = 0; i < numGPUs; i++)
    {   
        cudaErrorCheck(hipSetDevice(gpus[i]));
        cudaErrorCheck(hipStreamCreate(&stream1[i]));
        cudaErrorCheck(hipStreamCreate(&stream2[i]));
        // memory allocation on the GPU.
        cudaErrorCheck(hipMalloc(d_p + i, size * sizeof(double)));
        cudaErrorCheck(hipMalloc(d_A + i, num_rows * size * sizeof(double)));
        cudaErrorCheck(hipMalloc(d_Ap + i, numGPUs * num_rows * sizeof(double)));
        cudaErrorCheck(hipMalloc(d_diagA + i, size * sizeof(double)));

        // each GPU gets a subset of the matrix A
        if (i < numGPUs - 1)
        {
            cudaErrorCheck(hipMemcpyAsync(d_A[i], A + i * num_rows * size, num_rows * size * sizeof(double), hipMemcpyHostToDevice, stream2[i]));
        }
        else
        {
            cudaErrorCheck(hipMemcpyAsync(d_A[i], A + i * num_rows * size, (num_rows - unused_rows) * size * sizeof(double), hipMemcpyHostToDevice, stream2[i]));
        }
    }   

    // if the matrix is not divisible by 4, fill the empty rows from last GPU with zeros
    if (not_divisible)
    {
        cudaErrorCheck(hipMemsetAsync(d_A[numGPUs - 1] + (num_rows - unused_rows) * size, 0, unused_rows * size * sizeof(double), stream1[numGPUs - 1]));
    }
    
    cudaErrorCheck(hipSetDevice(gpus[0]));
    cudaErrorCheck(hipMemcpy(d_b, b, size * sizeof(double), hipMemcpyHostToDevice));

    dot <<<numBlocks, numThreads>>> (d_b, d_b, d_bb, size);
    for (int i = 0; i < numGPUs; i++)
    {
        hipSetDevice(gpus[i]);
        nccl_extract_diagonal <<<numBlocks, numThreads, 0, stream2[i]>>> (d_A[i], d_diagA[i] + i * num_rows, num_rows, size, unused_rows, i, numGPUs);
    }
    
    ncclGroupStart();
    for(int i = 0; i < numGPUs; i++) 
    {
        cudaErrorCheck(hipSetDevice(gpus[i]));
        ncclErrorCheck(ncclAllGather(d_diagA[i] + i * num_rows, d_diagA[i], num_rows, ncclDouble, comms[i], stream2[i]));
    }
    ncclGroupEnd();
    hipSetDevice(gpus[0]);
    apply_preconditioner <<<numBlocks, numThreads, 0, stream2[0]>>> (d_b, d_z, d_diagA[0], size);
    dot <<<numBlocks, numThreads, 0, stream2[0]>>> (d_b, d_z, d_rr_new, size);
    initialization <<<numBlocks, numThreads, 0, stream1[0]>>> (d_x, d_b, d_r, size);
    cudaErrorCheck(hipMemcpyAsync(d_p[0], d_z, size * sizeof(double), hipMemcpyDeviceToDevice, stream2[0]));
    cudaErrorCheck(hipMemcpyAsync(&rr, d_rr_new, sizeof(double), hipMemcpyDeviceToHost,stream2[0]));
    cudaErrorCheck(hipMemcpy(&bb, d_bb, sizeof(double), hipMemcpyDeviceToHost));

    hipblasHandle_t handle[numGPUs];
    for(int i = 0; i < numGPUs; i++)
    {
        cudaErrorCheck(hipSetDevice(gpus[i]));
        cublasErrorCheck(hipblasCreate(&handle[i]));
        cublasErrorCheck(hipblasSetStream(handle[i], stream1[i])); // link the cuda stream to the cublas handle
    }

    for (int i = 0; i < numGPUs; i++)
    {
        hipSetDevice(gpus[i]);
        hipStreamSynchronize(stream2[i]);
    }
    cudaErrorCheck(hipSetDevice(gpus[0]));


    // MAIN LOOP
    for(num_iters = 1; num_iters <= max_iters; num_iters++)
    {   
        // broadcast d_p to all GPUs
        ncclGroupStart();
        for(int i = 0; i < numGPUs; i++) 
        {
            cudaErrorCheck(hipSetDevice(gpus[i]));
            ncclErrorCheck(ncclBroadcast(d_p[0], d_p[i], size, ncclDouble, 0, comms[i], stream1[i]));
        }
        ncclGroupEnd();

        // each GPU computes gemv on a subset of the matrix A
        for (int i = 0; i < numGPUs; i++)
        {   
            hipSetDevice(gpus[i]);
            cublasErrorCheck(hipblasDgemv(handle[i], HIPBLAS_OP_T, size, num_rows, &gemv_alpha, d_A[i], size, d_p[i], 1, &gemv_beta, d_Ap[i] + i * num_rows, 1));
        }
        
        // allgather the results of gemv
        ncclErrorCheck(ncclGroupStart());   
        for (int i = 0; i < numGPUs; i++)
        {
            hipSetDevice(gpus[i]);
            ncclErrorCheck(ncclAllGather(d_Ap[i] + i * num_rows, d_Ap[i], num_rows, ncclDouble, comms[i], stream1[i]));
        }
        ncclErrorCheck(ncclGroupEnd());

        // synchronize over all GPUs
        for (int i = 0; i < numGPUs; i++)
        {
            hipSetDevice(gpus[i]);
            hipStreamSynchronize(stream1[i]);
        }
        cudaErrorCheck(hipSetDevice(gpus[0]));

        dot <<<numBlocks, numThreads>>> (d_p[0], d_Ap[0], d_pAp, size);
        cudaErrorCheck(hipMemcpy(&pAp, d_pAp, sizeof(double), hipMemcpyDeviceToHost));
        alpha = rr / pAp;
        axpby <<<numBlocks, numThreads>>> (-alpha, d_Ap[0], 1.0, d_r, size); 
        hipMemsetAsync(d_pAp, 0, sizeof(double)); // reset dot product to zero, done in parallel with stream2
        hipMemsetAsync(d_rr_new, 0, sizeof(double));
        hipStreamSynchronize(stream2[0]); // ensure that axbpy on x from the previous iteration has terminated
        axpby <<<numBlocks, numThreads, 0, stream2[0]>>> (alpha, d_p[0], 1.0, d_x, size); // x is not needed until the next iteration and is only get called by this kernel
        apply_preconditioner <<<numBlocks, numThreads>>> (d_r, d_z, d_diagA[0], size);
        dot <<<numBlocks, numThreads>>> (d_r, d_z, d_rr_new, size);
        cudaErrorCheck(hipMemcpy(&rr_new, d_rr_new, sizeof(double), hipMemcpyDeviceToHost));
        beta = rr_new / rr;
        xpby <<<numBlocks, numThreads>>> (d_z, beta, d_p[0], size); // this can be done after beta is calculated
        rr = rr_new;
        if(std::sqrt(rr / bb) < rel_error) { break; }
    }
    
    // copy the solution back to the host
    cudaErrorCheck(hipMemcpyAsync(x, d_x, size * sizeof(double), hipMemcpyDeviceToHost,stream2[0]));

    // cleaning up
    cudaErrorCheck(hipFree(d_x));
    cudaErrorCheck(hipFree(d_b));
    cudaErrorCheck(hipFree(d_r));
    cudaErrorCheck(hipFree(d_bb));
    cudaErrorCheck(hipFree(d_pAp));
    cudaErrorCheck(hipFree(d_rr_new));
    cudaErrorCheck(hipFree(d_z));
    for(int i = 0; i < numGPUs; i++)
    {   
        cudaErrorCheck(hipFree(d_p[i]));
        cudaErrorCheck(hipFree(d_A[i]));
        cudaErrorCheck(hipFree(d_Ap[i]));
        cudaErrorCheck(hipFree(d_diagA[i]));
        cudaErrorCheck(hipStreamDestroy(stream1[i]));
        cudaErrorCheck(hipStreamDestroy(stream2[i]));
        ncclErrorCheck(ncclCommDestroy(comms[i]));
        cublasErrorCheck(hipblasDestroy(handle[i]));
    }

    if(num_iters <= max_iters)
    {
        printf("Converged in %d iterations, relative error is %e\n", num_iters, std::sqrt(rr / bb));
    }
    else
    {
        printf("Did not converge in %d iterations, relative error is %e\n", max_iters, std::sqrt(rr / bb));
    }
}





void conjugate_gradients(const double * A, const double * b, double * x, size_t size, int max_iters, double rel_error)
{   
    double alpha, beta, bb, rr, rr_new;
    double pAp;
    const double gemv_alpha = 1.0;
    const double gemv_beta = 0.0;
    double * d_A = new double[size * size];
    double * d_diagA = new double[size];
    double * d_Ap = new double[size];
    double * d_x = new double[size];
    double * d_b = new double[size];
    double * d_r = new double[size];
    double * d_p = new double[size];
    double * d_z = new double[size];
    double * d_bb = new double;
    double * d_rr_new = new double;
    double * d_pAp = new double;
    int num_iters;

    int numBlocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int numThreads = BLOCK_SIZE;
    
    hipStream_t stream1, stream2, stream3;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // memory allocation on the GPU.
    cudaErrorCheck(hipMalloc((void**)&d_x, size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_b, size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_r, size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_p, size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_z, size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_bb, sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_rr_new, sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_A, size * size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_diagA, size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_Ap, size * sizeof(double)));
    cudaErrorCheck(hipMalloc((void**)&d_pAp, sizeof(double)));

    // memory copy from the CPU to the GPU, copy of matrix A is done in parallel with initialization and dot product.
    cudaErrorCheck(hipMemcpyAsync(d_A, A, size * size * sizeof(double), hipMemcpyHostToDevice, stream1));
    cudaErrorCheck(hipMemcpy(d_b, b, size * sizeof(double), hipMemcpyHostToDevice));
    
    dot <<<numBlocks, numThreads>>> (d_b, d_b, d_bb, size);
    extract_diagonal <<<numBlocks, numThreads, 0, stream1>>> (d_A, d_diagA, size);
    apply_preconditioner <<<numBlocks, numThreads, 0, stream1>>> (d_b, d_z, d_diagA, size);
    initialization <<<numBlocks, numThreads, 0, stream2>>> (d_x, d_b, d_r, size);
    cudaErrorCheck(hipMemcpyAsync(d_p, d_z, size * sizeof(double), hipMemcpyDeviceToDevice, stream1));
    cudaErrorCheck(hipMemcpyAsync(&bb, d_bb, sizeof(double), hipMemcpyDeviceToHost,stream3));
    dot <<<numBlocks, numThreads, 0, stream1>>> (d_b, d_z, d_rr_new, size); 
    cudaErrorCheck(hipMemcpy(&rr, d_rr_new, sizeof(double), hipMemcpyDeviceToHost));
    cudaErrorCheck(hipDeviceSynchronize());

    for(num_iters = 1; num_iters <= max_iters; num_iters++)
    {
        hipblasDgemv(handle, HIPBLAS_OP_N, size, size, &gemv_alpha, d_A, size, d_p, 1, &gemv_beta, d_Ap, 1);
        dot <<<numBlocks, numThreads>>> (d_p, d_Ap, d_pAp, size);
        cudaErrorCheck(hipMemcpy(&pAp, d_pAp, sizeof(double), hipMemcpyDeviceToHost));
        alpha = rr / pAp;
        axpby <<<numBlocks, numThreads>>> (-alpha, d_Ap, 1.0, d_r, size); 
        hipMemsetAsync(d_pAp, 0, sizeof(double)); // reset dot product to zero, done in parallel with stream1
        hipMemsetAsync(d_rr_new, 0, sizeof(double));
        hipStreamSynchronize(stream1); // ensure that axbpy on x from the previous iteration has terminated
        axpby <<<numBlocks, numThreads, 0, stream1>>> (alpha, d_p, 1.0, d_x, size); // x is not needed until the next iteration and is only get called by this kernel
        apply_preconditioner <<<numBlocks, numThreads>>> (d_r, d_z, d_diagA, size);
        dot <<<numBlocks, numThreads>>> (d_r, d_z, d_rr_new, size);
        cudaErrorCheck(hipMemcpy(&rr_new, d_rr_new, sizeof(double), hipMemcpyDeviceToHost));
        beta = rr_new / rr;
        xpby <<<numBlocks, numThreads>>> (d_z, beta, d_p, size); // this can be done after beta is calculated
        rr = rr_new;
        if(std::sqrt(rr / bb) < rel_error) { break; }
    }

    cudaErrorCheck(hipMemcpyAsync(x, d_x, size * sizeof(double), hipMemcpyDeviceToHost,stream1));

    // cleaning up
    cudaErrorCheck(hipFree(d_x));
    cudaErrorCheck(hipFree(d_b));
    cudaErrorCheck(hipFree(d_r));
    cudaErrorCheck(hipFree(d_p));
    cudaErrorCheck(hipFree(d_z));
    cudaErrorCheck(hipFree(d_A));
    cudaErrorCheck(hipFree(d_diagA));
    cudaErrorCheck(hipFree(d_Ap));
    cudaErrorCheck(hipFree(d_bb));
    cudaErrorCheck(hipFree(d_pAp));
    cudaErrorCheck(hipFree(d_rr_new));
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    hipblasDestroy(handle);

    if(num_iters <= max_iters)
    {
        printf("Converged in %d iterations, relative error is %e\n", num_iters, std::sqrt(rr / bb));
    }
    else
    {
        printf("Did not converge in %d iterations, relative error is %e\n", max_iters, std::sqrt(rr / bb));
    }
}



int main(int argc, char ** argv)
{
    printf("Usage: ./random_matrix input_file_matrix.bin input_file_rhs.bin output_file_sol.bin max_iters rel_error\n");
    printf("All parameters are optional and have default values\n");
    printf("\n");

    const char * input_file_matrix = "io/matrix.bin";
    const char * input_file_rhs = "io/rhs.bin";
    const char * output_file_sol = "io/sol.bin";
    int max_iters = 1000;
    double rel_error = 1e-9;

    if(argc > 1) input_file_matrix = argv[1];
    if(argc > 2) input_file_rhs = argv[2];
    if(argc > 3) output_file_sol = argv[3];
    if(argc > 4) max_iters = atoi(argv[4]);
    if(argc > 5) rel_error = atof(argv[5]);

    printf("Command line arguments:\n");
    printf("  input_file_matrix: %s\n", input_file_matrix);
    printf("  input_file_rhs:    %s\n", input_file_rhs);
    printf("  output_file_sol:   %s\n", output_file_sol);
    printf("  max_iters:         %d\n", max_iters);
    printf("  rel_error:         %e\n", rel_error);
    printf("\n");



    double * matrix;
    double * rhs;
    size_t size;

    {
        printf("Reading matrix from file ...\n");
        size_t matrix_rows;
        size_t matrix_cols;
        bool success_read_matrix = read_matrix_from_file(input_file_matrix, &matrix, &matrix_rows, &matrix_cols);
        if(!success_read_matrix)
        {
            fprintf(stderr, "Failed to read matrix\n");
            return 1;
        }
        printf("Done\n");
        printf("\n");

        printf("Reading right hand side from file ...\n");
        size_t rhs_rows;
        size_t rhs_cols;
        bool success_read_rhs = read_matrix_from_file(input_file_rhs, &rhs, &rhs_rows, &rhs_cols);
        if(!success_read_rhs)
        {
            fprintf(stderr, "Failed to read right hand side\n");
            return 2;
        }
        printf("Done\n");
        printf("\n");

        if(matrix_rows != matrix_cols)
        {
            fprintf(stderr, "Matrix has to be square\n");
            return 3;
        }
        if(rhs_rows != matrix_rows)
        {
            fprintf(stderr, "Size of right hand side does not match the matrix\n");
            return 4;
        }
        if(rhs_cols != 1)
        {
            fprintf(stderr, "Right hand side has to have just a single column\n");
            return 5;
        }

        size = matrix_rows;
    }

    printf("Solving the system ...\n");
    double * sol = new double[size];

    start = clock();

    // if the matrix is big use the implementation with NCCL (multiple GPUs, each with a subset of the matrix A)
    if (size <= 65536)
        conjugate_gradients(matrix, rhs, sol, size, max_iters, rel_error);
    else
        nccl_conjugate_gradients(matrix, rhs, sol, size, max_iters, rel_error);

    end = clock();

    printf("Done\n");
    printf("\n");

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Execution time: %f seconds\n", cpu_time_used);
    printf("\n");

    printf("Writing solution to file ...\n");
    bool success_write_sol = write_matrix_to_file(output_file_sol, sol, size, 1);
    if(!success_write_sol)
    {
        fprintf(stderr, "Failed to save solution\n");
        return 6;
    }
    printf("Done\n");
    printf("\n");

    #ifdef APPROX
    print_approx_solution(sol, size, 1);
    #endif

    delete[] matrix;
    delete[] rhs;
    delete[] sol;

    printf("Finished successfully\n");

    return 0;
}
